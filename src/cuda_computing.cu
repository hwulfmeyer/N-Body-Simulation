#include "hip/hip_runtime.h"



#include "cuda_computing.cuh"

#define NUM_THREADS_PER_BLOCK 128
#define NUM_BODIES_PER_THREAD 4

namespace Device {
	// CUDA global constants
	__device__ __constant__
		float EPSILON2;
	__device__ __constant__
		float DTGRAVITY;
	__device__ __constant__
		int NBODIES;
	__device__ __constant__
		int NTHREADS;

	// array of masses
	float *masses;
	// array of velocities
	float3 *velocities;
	// array of positions
	float3 *positions;

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies
	// NOTES: try not using EPSILON for calculations
	// NOTES: more than one particle in one thread
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__
		float3
		bodyBodyInteraction(float3 myPos, float3 othPos, float mass_oth, float3 velo) {
		float3 dir;
		//3 FLOP
		dir.x = othPos.x - myPos.x;
		dir.y = othPos.y - myPos.y;
		dir.z = othPos.z - myPos.z;
		// 6 FLOP
		float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPSILON2;
		// 4 FLOP
		float partForce = mass_oth / sqrtf(distSqr*distSqr*distSqr);
		// 6 FLOP
		velo.x += dir.x * partForce;
		velo.y += dir.y * partForce;
		velo.z += dir.z * partForce;
		// in total 19 FLOP per body body Interaction
		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// naive kernel computing velocities
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		computeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
		if (tidx < NBODIES) {
			float3 myPos = positions[tidx];
			float3 myVelo = velocities[tidx];
			for (unsigned int k = 0; k < NBODIES; ++k)
			{
				myVelo = bodyBodyInteraction(myPos, positions[k], masses[k], myVelo);
			}

			myPos.x += myVelo.x * DTGRAVITY;
			myPos.y += myVelo.y * DTGRAVITY;
			myPos.z += myVelo.z * DTGRAVITY;

			positions[tidx] = myPos;
			velocities[tidx] = myVelo;
		}
	}


	////////////////////////////////////////////////////////////////////////////////////////////////////
	// naive kernel computing velocities + doing two body calculations at once
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		taoComputeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidA = blockIdx.x * blockDim.x * 2 + threadIdx.x * 2;
		unsigned int tidB = tidA + 1;

		if (tidA < NBODIES) {
			if (tidB < NBODIES) {
				float3 myPosA = positions[tidA];
				float3 myPosB = positions[tidB];
				float3 myVeloA = velocities[tidA];
				float3 myVeloB = velocities[tidB];
				for (unsigned int k = 0; k < NBODIES; ++k)
				{
					float3 curPos = positions[k];
					float curMass = masses[k];
					myVeloA = bodyBodyInteraction(myPosA, curPos, curMass, myVeloA);
					myVeloB = bodyBodyInteraction(myPosB, curPos, curMass, myVeloB);
				}
				myPosA.x += myVeloA.x * DTGRAVITY;
				myPosA.y += myVeloA.y * DTGRAVITY;
				myPosA.z += myVeloA.z * DTGRAVITY;
				myPosB.x += myVeloB.x * DTGRAVITY;
				myPosB.y += myVeloB.y * DTGRAVITY;
				myPosB.z += myVeloB.z * DTGRAVITY;

				positions[tidA] = myPosA;
				positions[tidB] = myPosB;
				velocities[tidA] = myVeloA;
				velocities[tidB] = myVeloB;
			}
			else {
				float3 myPos = positions[tidA];
				float3 myVelo = velocities[tidA];
				for (unsigned int k = 0; k < NBODIES; ++k)
				{
					myVelo = bodyBodyInteraction(myPos, positions[k], masses[k], myVelo);
				}
				myPos.x += myVelo.x * DTGRAVITY;
				myPos.y += myVelo.y * DTGRAVITY;
				myPos.z += myVelo.z * DTGRAVITY;

				positions[tidA] = myPos;
				velocities[tidA] = myVelo;
			}
		}
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// naive kernel computing velocities + doing NUM_BODIES_PER_THREAD body calculations at once
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		xaoComputeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tids[NUM_BODIES_PER_THREAD];
		tids[0] = blockIdx.x * blockDim.x * NUM_BODIES_PER_THREAD + threadIdx.x * NUM_BODIES_PER_THREAD;

		if (tids[0] < NBODIES) {
			unsigned int i_max = 0;
			for (unsigned int i = 1; i < NUM_BODIES_PER_THREAD && tids[i_max] < NBODIES; ++i, ++i_max) {
				tids[i] = tids[i - 1] + 1;
			}
			float3 myPos[NUM_BODIES_PER_THREAD];
			float3 myVelo[NUM_BODIES_PER_THREAD];

			//in our tids array no tids are out of bounds
			if (i_max + 1 == NUM_BODIES_PER_THREAD) {

				for (int i = 0; i < NUM_BODIES_PER_THREAD; ++i) {
					myPos[i] = positions[tids[i]];
					myVelo[i] = velocities[tids[i]];
				}

				for (unsigned int k = 0; k < NBODIES; ++k)
				{
					for (int i = 0; i < NUM_BODIES_PER_THREAD; ++i) {
						myVelo[i] = bodyBodyInteraction(myPos[i], positions[k], masses[k], myVelo[i]);
					}
				}

				for (int i = 0; i < NUM_BODIES_PER_THREAD; ++i) {
					myPos[i].x += myVelo[i].x * DTGRAVITY;
					myPos[i].y += myVelo[i].y * DTGRAVITY;
					myPos[i].z += myVelo[i].z * DTGRAVITY;

					positions[tids[i]] = myPos[i];
					velocities[tids[i]] = myVelo[i];
				}
			}
			// we have got tids that are out of bounds, take i_max+1 instead as max value
			else {

				for (int i = 0; i < i_max; ++i) {
					myPos[i] = positions[tids[i]];
					myVelo[i] = velocities[tids[i]];
				}

				for (unsigned int k = 0; k < NBODIES; ++k)
				{
					for (int i = 0; i < i_max; ++i) {
						myVelo[i] = bodyBodyInteraction(myPos[i], positions[k], masses[k], myVelo[i]);
					}
				}

				for (int i = 0; i < i_max; ++i) {
					myPos[i].x += myVelo[i].x * DTGRAVITY;
					myPos[i].y += myVelo[i].y * DTGRAVITY;
					myPos[i].z += myVelo[i].z * DTGRAVITY;

					positions[tids[i]] = myPos[i];
					velocities[tids[i]] = myVelo[i];
				}
			}
		}
	}


	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies [v1.SHARED MEMORY]
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__
		float3
		smV1BodyBodyInteraction(float3 myPos, float4 othPos, float3 velo) {
		float3 dir;
		//3 FLOP
		dir.x = othPos.x - myPos.x;
		dir.y = othPos.y - myPos.y;
		dir.z = othPos.z - myPos.z;
		// 6 FLOP
		float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPSILON2;
		// 4 FLOP
		float partForce = othPos.w / sqrtf(distSqr*distSqr*distSqr);
		// 6 FLOP
		velo.x += dir.x * partForce;
		velo.y += dir.y * partForce;
		velo.z += dir.z * partForce;
		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// kernel v.SHARED
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		smV1ComputeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
		extern __shared__ float4 smPos[];

		if (tidx < NBODIES) {
			float3 myPos = positions[tidx];
			float3 myVelo = velocities[tidx];
			// each loop step copies NUM_THREADS_PER_BLOCK values into shared memory
			// hence we have to do it gridDim.x = NBODIES/NUM_THREADS_PER_BLOCK times to get to each body
			for (int curTileIdx = 0; curTileIdx < gridDim.x; curTileIdx++) {
				int idx = curTileIdx * blockDim.x + threadIdx.x;
				smPos[threadIdx.x].x = positions[idx].x;
				smPos[threadIdx.x].y = positions[idx].y;
				smPos[threadIdx.x].z = positions[idx].z;
				smPos[threadIdx.x].w = masses[idx];
				__syncthreads();
				//compute interactions in our current sharedMemory

				for (unsigned int i = 0; i < NTHREADS; i++)
				{
					myVelo = smV1BodyBodyInteraction(myPos, smPos[i], myVelo);
				}
				__syncthreads();
			}

			myPos.x += myVelo.x * DTGRAVITY;
			myPos.y += myVelo.y * DTGRAVITY;
			myPos.z += myVelo.z * DTGRAVITY;

			__syncthreads();
			positions[tidx] = myPos;
			velocities[tidx] = myVelo;
		}
	}


	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies [v2.SHARED MEMORY]
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__
		float3
		smV2BodyBodyInteraction(float3 myPos, float4 othPos, float3 velo) {
		float3 dir;
		//3 FLOP
		dir.x = othPos.x - myPos.x;
		dir.y = othPos.y - myPos.y;
		dir.z = othPos.z - myPos.z;
		// 6 FLOP
		float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPSILON2;
		// 4 FLOP
		float partForce = rsqrtf(distSqr*distSqr*distSqr);
		partForce *= othPos.w;
		// 6 FLOP
		velo.x += dir.x * partForce;
		velo.y += dir.y * partForce;
		velo.z += dir.z * partForce;
		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// kernel v.SHARED + LOOP UNROLL + rsqrtf()
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		smV3ComputeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
		extern __shared__ float4 smPos[];

		if (tidx < NBODIES) {
			float3 myPos = positions[tidx];
			float3 myVelo = velocities[tidx];
			// each loop step copies NUM_THREADS_PER_BLOCK values into shared memory
			// hence we have to do it gridDim.x = NBODIES/NUM_THREADS_PER_BLOCK times to get to each body
			for (int curTileIdx = 0; curTileIdx < gridDim.x; curTileIdx++) {
				int idx = curTileIdx * blockDim.x + threadIdx.x;
				smPos[threadIdx.x].x = positions[idx].x;
				smPos[threadIdx.x].y = positions[idx].y;
				smPos[threadIdx.x].z = positions[idx].z;
				smPos[threadIdx.x].w = masses[idx];
				__syncthreads();
				//compute interactions in our current sharedMemory

				for (unsigned int i = 0; i < NUM_THREADS_PER_BLOCK; i++)
				{
					myVelo = smBodyBodyInteraction(myPos, smPos[i], myVelo);
				}
				__syncthreads();
			}

			myPos.x += myVelo.x * DTGRAVITY;
			myPos.y += myVelo.y * DTGRAVITY;
			myPos.z += myVelo.z * DTGRAVITY;

			__syncthreads();
			positions[tidx] = myPos;
			velocities[tidx] = myVelo;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// constructor, copies all the bodies into this class
////////////////////////////////////////////////////////////////////////////////////////////////////
Cuda_Computing::Cuda_Computing(std::vector<Body> &bodies) : N(bodies.size()) {
	this->positions = new float3[N];
	this->masses = new float[N];
	this->velocities = new float3[N];

	for (unsigned int i = 0; i < N; ++i)
	{
		positions[i].x = bodies[i].position.x;
		positions[i].y = bodies[i].position.y;
		positions[i].z = bodies[i].position.z;

		masses[i] = bodies[i].mass;

		velocities[i].x = bodies[i].velocity.x;
		velocities[i].y = bodies[i].velocity.y;
		velocities[i].z = bodies[i].velocity.z;
	}

	std::cerr << "Cuda_Computing::Cuda_Computing() - Copying of " << N << " bodies done." << std::endl;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// initializes device, detects hardware, number of threads per block
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initDevice() {
	//check execution environment
	int deviceCount = 0;
	int device_handle = 0;
	errorCheckCuda(hipGetDeviceCount(&deviceCount));

	if (0 == deviceCount) {
		std::cerr << "initDevice() : No CUDA device found." << std::endl;
		return false;
	}

	if (deviceCount > 1) {
		std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
	}

	// set the device
	errorCheckCuda(hipSetDevice(device_handle));

	hipDeviceProp_t device_props;
	errorCheckCuda(hipGetDeviceProperties(&device_props, device_handle));
	//std::cerr << "Max CC: " << device_props.major << "   Min CC: " << device_props.minor << std::endl;

	// determine thread layout
	// num of threads on 1 block, thread layout per block
	blockSize = dim3(NUM_THREADS_PER_BLOCK, 1, 1);
	int numBlocks = N / NUM_THREADS_PER_BLOCK;
	if (0 != N % blockSize.x) numBlocks++;
	// number of blocks, block layout on grid
	gridSize = dim3(numBlocks, 1, 1);

	//determine thread layout when doing 2 body calculations per thread
	int numBlocksTAO = N / (NUM_THREADS_PER_BLOCK * 2);
	if (0 != N % (blockSize.x * 2)) numBlocksTAO++;
	// number of blocks, block layout on grid
	gridSizeTAO = dim3(numBlocksTAO, 1, 1);

	//determine thread layout when doing NUM_BODIES_PER_THREAD body calculations per thread
	int numBlocksXAO = N / (NUM_THREADS_PER_BLOCK * NUM_BODIES_PER_THREAD);
	if (0 != N % (blockSize.x * NUM_BODIES_PER_THREAD)) numBlocksXAO++;
	// number of blocks, block layout on grid
	gridSizeXAO = dim3(numBlocksXAO, 1, 1);

	std::cerr << "num blocks = " << gridSize.x << " :: "
		<< "threads per Block = " << blockSize.x << " :: "
		<< "num blocks tao = " << gridSizeTAO.x << " :: "
		<< "num blocks xao = " << gridSizeXAO.x << std::endl;

	float dtG = G*DT;
	int nTh = NUM_THREADS_PER_BLOCK;

	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::EPSILON2), &EPS2, sizeof(float), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::DTGRAVITY), &dtG, sizeof(float), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::NBODIES), &N, sizeof(int), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::NTHREADS), &blockSize.x, sizeof(int), 0, hipMemcpyHostToDevice));
	return true;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// creating vertexBuffer for openGL/cuda inop
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initVertexBuffer() {
	// allocate & register the vertexbuffer
	hipGraphicsResource *cuda_vbo_resources[3];
	GLuint vao;
	GLuint vbo_pos;
	GLuint vbo_mass;
	GLuint vbo_velos;

	// create a vertex array of our device pointer for opengl/cuda inop
	glGenVertexArrays(3, &vao);
	glBindVertexArray(vao);

	glGenBuffers(1, &vbo_pos);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_pos);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float3), positions, GL_DYNAMIC_COPY); 	// buffer data with our positions
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(0);

	glGenBuffers(1, &vbo_mass);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_mass);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float), masses, GL_DYNAMIC_COPY);
	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(1);

	glGenBuffers(1, &vbo_velos);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_velos);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float3), velocities, GL_DYNAMIC_COPY);
	glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(2);

	glBindVertexArray(vao);

	//cudaGLRegisterBufferObject(vbo); ///deprecated
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[0], vbo_pos, cudaGraphicsMapFlagsNone));
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[1], vbo_mass, cudaGraphicsMapFlagsNone));
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[2], vbo_velos, cudaGraphicsMapFlagsNone));

	// Map the buffer to CUDA
	//cudaGLMapBufferObject(&vptr, vbo); ///deprecated
	errorCheckCuda(hipGraphicsMapResources(3, cuda_vbo_resources));
	size_t numBytes;
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::positions, &numBytes, cuda_vbo_resources[0]));
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::masses, &numBytes, cuda_vbo_resources[1]));
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::velocities, &numBytes, cuda_vbo_resources[2]));

	// Unmap the buffer
	//cudaGLUnmapBufferObject(vbo); /// deprecated
	errorCheckCuda(hipGraphicsUnmapResources(3, cuda_vbo_resources));
	return true;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// kernel entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
float
Cuda_Computing::computeNewPositions() {
	float time;
	hipEvent_t start, stop;
	errorCheckCuda(hipEventCreate(&start));
	errorCheckCuda(hipEventCreate(&stop));
	errorCheckCuda(hipEventRecord(start, 0));

	//Device::computeVelocities << < gridSize, blockSize
	//	>> > (Device::positions, Device::masses, Device::velocities);

	Device::smV3ComputeVelocities << < gridSize, blockSize, sizeof(float4)*NUM_THREADS_PER_BLOCK
		>> > (Device::positions, Device::masses, Device::velocities);

	//Device::taoComputeVelocities << < gridSizeTAO, blockSize
	//	>> > (Device::positions, Device::masses, Device::velocities);

	//Device::xaoComputeVelocities << < gridSizeXAO, blockSize
	//	>> > (Device::positions, Device::masses, Device::velocities);

	//errorCheckCuda(hipPeekAtLastError());
	errorCheckCuda(hipDeviceSynchronize());
	errorCheckCuda(hipEventRecord(stop, 0));
	errorCheckCuda(hipEventSynchronize(stop));
	errorCheckCuda(hipEventElapsedTime(&time, start, stop));

	return time;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// returns the number of bodies 
////////////////////////////////////////////////////////////////////////////////////////////////////
size_t
Cuda_Computing::getSize() const {
	return N;
}
