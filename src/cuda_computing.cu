#include "hip/hip_runtime.h"



#include "cuda_computing.cuh"

#define NUM_THREADS_PER_BLOCK 512

namespace Device {
	// CUDA global constants
	__device__ __constant__
		float EPSILON2;
	__device__ __constant__
		float DTGRAVITY;
	__device__ __constant__
		int NBODIES;
	__device__ __constant__
		int NTHREADS;

	// array of masses
	float *masses;
	// array of velocities
	float3 *velocities;
	// array of positions
	float3 *positions;

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies
	// NOTES: try not using EPSILON for calculations
	// NOTES: more than one particle in one thread
	// NOTES: loop unrolling
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__
		float3
		bodyBodyInteraction(float3 myPos, float3 othPos, float mass_oth, float3 velo) {
		float3 dir;
		//3 FLOP
		dir.x = othPos.x - myPos.x;
		dir.y = othPos.y - myPos.y;
		dir.z = othPos.z - myPos.z;
		// 6 FLOP
		float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPSILON2;
		// 4 FLOP
		float partForce = mass_oth / sqrtf(distSqr*distSqr*distSqr);
		// 6 FLOP
		velo.x += dir.x * partForce;
		velo.y += dir.y * partForce;
		velo.z += dir.z * partForce;
		// in total 19 FLOP per body body Interaction
		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// kernel computing velocities
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		computeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
		if (tidx < NBODIES) {
			float3 myPos = positions[tidx];
			float3 myVelo = velocities[tidx];
			for (unsigned int k = 0; k < NBODIES; ++k)
			{
				myVelo = bodyBodyInteraction(myPos, positions[k], masses[k], myVelo);
			}
			
			myPos.x += myVelo.x * DTGRAVITY;
			myPos.y += myVelo.y * DTGRAVITY;
			myPos.z += myVelo.z * DTGRAVITY;
	
			positions[tidx] = myPos;
			velocities[tidx] = myVelo;
		}
	}


	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies [SHARED MEMORY]
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__
		float3
		smBodyBodyInteraction(float3 myPos, float4 othPos, float3 velo) {
		float3 dir;
		//3 FLOP
		dir.x = othPos.x - myPos.x;
		dir.y = othPos.y - myPos.y;
		dir.z = othPos.z - myPos.z;
		// 6 FLOP
		float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPSILON2;
		// 4 FLOP
		float partForce = othPos.w / sqrtf(distSqr*distSqr*distSqr);
		// 6 FLOP
		velo.x += dir.x * partForce;
		velo.y += dir.y * partForce;
		velo.z += dir.z * partForce;
		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// physics calculations between bodies in a tile [SHARED MEMORY]
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__ float3 smTileCalculation(float3 myPos, float3 velo)
	{
		extern __shared__ float4 smPos[];

		for (unsigned int i = 0; i < NTHREADS; i++)
		{
			velo = smBodyBodyInteraction(myPos, smPos[i], velo);
		}

		return velo;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	// kernel using shared Memory computing velocities
	// NVS4200M has 48KB of SM per SMP meaning 1 block on one SMP uses that much, 2 blocks split the 48KB...
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__
		void
		smComputeVelocities(float3 *positions, float* masses, float3 *velocities) {
		unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
		extern __shared__ float4 smPos[];

		if (tidx < NBODIES) {
			float3 myPos = positions[tidx];
			float3 myVelo = velocities[tidx];

			for (int i = 0, b_tile = 0; i < NBODIES; i += NTHREADS, b_tile++) {
				int idx = b_tile * blockDim.x + threadIdx.x;
				smPos[threadIdx.x].x = positions[idx].x;
				smPos[threadIdx.x].y = positions[idx].y;
				smPos[threadIdx.x].z = positions[idx].z;
				smPos[threadIdx.x].w = masses[idx];
				__syncthreads();
				myVelo = smTileCalculation(myPos, myVelo);
				__syncthreads();
			}
			myPos.x += myVelo.x * DTGRAVITY;
			myPos.y += myVelo.y * DTGRAVITY;
			myPos.z += myVelo.z * DTGRAVITY;

			positions[tidx] = myPos;
			velocities[tidx] = myVelo;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// constructor, copies all the bodies into this class
////////////////////////////////////////////////////////////////////////////////////////////////////
Cuda_Computing::Cuda_Computing(std::vector<Body> &bodies) : N(bodies.size()) {
	this->positions = new float3[N];
	this->masses = new float[N];
	this->velocities = new float3[N];

	for (unsigned int i = 0; i < N; ++i)
	{
		positions[i].x = bodies[i].position.x;
		positions[i].y = bodies[i].position.y;
		positions[i].z = bodies[i].position.z;

		masses[i] = bodies[i].mass;

		velocities[i].x = bodies[i].velocity.x;
		velocities[i].y = bodies[i].velocity.y;
		velocities[i].z = bodies[i].velocity.z;
	}

	std::cerr << "Cuda_Computing::Cuda_Computing() - Copying of " << N << " bodies done." << std::endl;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// initializes device, detects hardware, number of threads per block
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initDevice() {
	//check execution environment
	int deviceCount = 0;
	int device_handle = 0;
	errorCheckCuda(hipGetDeviceCount(&deviceCount));

	if (0 == deviceCount) {
		std::cerr << "initDevice() : No CUDA device found." << std::endl;
		return false;
	}

	if (deviceCount > 1) {
		std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
	}

	// set the device
	errorCheckCuda(hipSetDevice(device_handle));

	hipDeviceProp_t device_props;
	errorCheckCuda(hipGetDeviceProperties(&device_props, device_handle));
	//std::cerr << "Max CC: " << device_props.major << "   Min CC: " << device_props.minor << std::endl;

	// determine thread layout
	// num of threads on 1 block, thread layout per block
	blockSize = dim3(NUM_THREADS_PER_BLOCK, 1, 1);
	int numBlocks = N / NUM_THREADS_PER_BLOCK;
	if (0 != N % blockSize.x) numBlocks++;
	// number of blocks, block layout on grid
	gridSize = dim3(numBlocks, 1, 1);

	std::cerr << "num blocks = " << numBlocks << " :: "
		<< "threads per Block = " << NUM_THREADS_PER_BLOCK << std::endl;

	float dtG = G*DT;
	int nTh = NUM_THREADS_PER_BLOCK;

	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::EPSILON2), &EPS2, sizeof(float), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::DTGRAVITY), &dtG, sizeof(float), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::NBODIES), &N, sizeof(int), 0, hipMemcpyHostToDevice));
	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::NTHREADS), &blockSize.x, sizeof(int), 0, hipMemcpyHostToDevice));
	return true;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// creating vertexBuffer for openGL/cuda inop
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initVertexBuffer() {
	// allocate & register the vertexbuffer
	hipGraphicsResource *cuda_vbo_resources[3];
	GLuint vao;
	GLuint vbo_pos;
	GLuint vbo_mass;
	GLuint vbo_velos;

	// create a vertex array of our device pointer for opengl/cuda inop
	glGenVertexArrays(3, &vao);
	glBindVertexArray(vao);

	glGenBuffers(1, &vbo_pos);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_pos);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float3), positions, GL_DYNAMIC_COPY); 	// buffer data with our positions
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(0);

	glGenBuffers(1, &vbo_mass);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_mass);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float), masses, GL_DYNAMIC_COPY);
	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(1);

	glGenBuffers(1, &vbo_velos);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_velos);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(float3), velocities, GL_DYNAMIC_COPY);
	glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(2);

	glBindVertexArray(vao);

	//cudaGLRegisterBufferObject(vbo); ///deprecated
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[0], vbo_pos, cudaGraphicsMapFlagsNone));
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[1], vbo_mass, cudaGraphicsMapFlagsNone));
	errorCheckCuda(hipGraphicsGLRegisterBuffer(&cuda_vbo_resources[2], vbo_velos, cudaGraphicsMapFlagsNone));

	// Map the buffer to CUDA
	//cudaGLMapBufferObject(&vptr, vbo); ///deprecated
	errorCheckCuda(hipGraphicsMapResources(3, cuda_vbo_resources));
	size_t numBytes;
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::positions, &numBytes, cuda_vbo_resources[0]));
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::masses, &numBytes, cuda_vbo_resources[1]));
	errorCheckCuda(hipGraphicsResourceGetMappedPointer((void**)&Device::velocities, &numBytes, cuda_vbo_resources[2]));

	// Unmap the buffer
	//cudaGLUnmapBufferObject(vbo); /// deprecated
	errorCheckCuda(hipGraphicsUnmapResources(3, cuda_vbo_resources));
	return true;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// kernel entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
void
Cuda_Computing::computeNewPositions() {
	// run kernel computing velocities
	//Device::computeVelocities << < gridSize, blockSize, 
	//	>> > (Device::positions, Device::masses, Device::velocities);
	Device::smComputeVelocities << < gridSize, blockSize, sizeof(float4)*NUM_THREADS_PER_BLOCK
		>> > (Device::positions, Device::masses, Device::velocities);
	//used only for error checking
	//errorCheckCuda(hipPeekAtLastError());
	errorCheckCuda(hipDeviceSynchronize());
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// returns the number of bodies 
////////////////////////////////////////////////////////////////////////////////////////////////////
size_t
Cuda_Computing::getSize() const {
	return N;
}
