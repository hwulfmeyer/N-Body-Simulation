#include "hip/hip_runtime.h"



#include "cuda_computing.cuh"

namespace Device {
	// array of coords
	glm::vec3 *positions;
	// array of masses
	float *masses;
	// array of velocities
	glm::vec3 *velocities;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// constructor, copies all the stuff to this class
////////////////////////////////////////////////////////////////////////////////////////////////////
Cuda_Computing::Cuda_Computing(std::vector<Body> &bodies) : size(bodies.size()) {
	this->positions = new glm::vec3[size];
	this->masses = new float[size];
	this->velocities = new glm::vec3[size];

	for (unsigned int i = 0; i < size; ++i)
	{
		positions[i] = bodies[i].position;

		masses[i] = bodies[i].mass;

		velocities[i] = bodies[i].velocity;
	}

	std::cout << "Cuda_Computing::Cuda_Computing() - Copying of " << size << " bodies done." << std::endl;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// destructor, deletes our dynamic arrays & frees memory on cuda device
////////////////////////////////////////////////////////////////////////////////////////////////////
Cuda_Computing::~Cuda_Computing() {
	// free dynamic arrays
	delete[] masses;
	delete[] velocities;

	//free arrays on cuda device
	checkErrorsCuda(hipFree(Device::positions));
	checkErrorsCuda(hipFree(Device::masses));
	checkErrorsCuda(hipFree(Device::velocities));
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// initializes device, detects hardware, number of threads per block
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initDevice() {
	//check execution environment
	int deviceCount = 0;
	int device_handle = 0;
	checkErrorsCuda(hipGetDeviceCount(&deviceCount));

	if (0 == deviceCount) {
		std::cerr << "initDevice() : No CUDA device found." << std::endl;
		return false;
	}

	if (deviceCount > 1) {
		std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
	}

	// set the device
	checkErrorsCuda(hipSetDevice(device_handle));

	hipDeviceProp_t device_props;
	checkErrorsCuda(hipGetDeviceProperties(&device_props, device_handle));
	//std::cout << "Max CC: " << device_props.major << "   Min CC: " << device_props.minor << std::endl;

	// determine max threads 
	unsigned int max_threads_per_block = device_props.maxThreadsPerBlock;
	unsigned int max_threads_per_block_sqrt = std::sqrt(max_threads_per_block);
	assert(max_threads_per_block_sqrt * max_threads_per_block_sqrt == max_threads_per_block);

	/* hard coding max threads cause of errors */
	max_threads_per_block_sqrt = 2;

	// determine thread layout
	num_threads_per_block = std::min(size, max_threads_per_block_sqrt);
	num_blocks = size / max_threads_per_block_sqrt;
	if (0 != size % max_threads_per_block) {
		num_blocks++;
	}
	std::cout << "num_blocks = " << num_blocks << " :: "
		<< "num_threads_per_block = " << num_threads_per_block << std::endl;

	// initialize memory
	Device::positions = nullptr;
	Device::masses = nullptr;
	Device::velocities = nullptr;
	return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// allocating device memory and copying memory to device
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initDeviceMemory() {
	Device::positions = this->positions;
	Device::masses = this->masses;
	Device::velocities = this->velocities;

	// allocate device memory
	checkErrorsCuda(hipMalloc(&Device::positions,
		size * sizeof(glm::vec3))
	);
	checkErrorsCuda(hipMalloc(&Device::masses,
		size * sizeof(float))
	);
	checkErrorsCuda(hipMalloc(&Device::velocities,
		size * sizeof(glm::vec3))
	);

	// copy device memory
	checkErrorsCuda(hipMemcpy(Device::positions, positions,
		size * sizeof(glm::vec3),
		hipMemcpyHostToDevice)
	);
	checkErrorsCuda(hipMemcpy(Device::masses, masses,
		size * sizeof(float),
		hipMemcpyHostToDevice)
	);
	checkErrorsCuda(hipMemcpy(Device::velocities, velocities,
		size * sizeof(glm::vec3),
		hipMemcpyHostToDevice)
	);

	return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// device physics calculations
////////////////////////////////////////////////////////////////////////////////////////////////////
__device__
glm::vec3
bodyBodyInteraction(glm::vec3 pos_body_cur, glm::vec3 pos_body_oth, float mass_oth, float EPS2) {
	glm::vec3 dir = pos_body_oth - pos_body_cur;
	float distSqr = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z + EPS2;

	float partForce = mass_oth / sqrt(distSqr*distSqr*distSqr);
	return dir * partForce;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// kernel
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void
device_computeForces(glm::vec3 *positions, float* masses, glm::vec3 *velocities, const float dtG, const int N, float EPS2) {
	
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N) {
		for (unsigned int k = 0; k < N; ++k)
		{
			velocities[tid] += bodyBodyInteraction(positions[tid], positions[k], masses[k], EPS2);
		}

		positions[tid] += dtG * velocities[tid];
	}
	
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// kernel entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
void
Cuda_Computing::computeForces(const float &dtG) {
	hipDeviceSynchronize();

	//run kernel
	device_computeForces <<< num_blocks, num_threads_per_block >> > (Device::positions, Device::masses, Device::velocities, dtG, size, EPS2);
	hipDeviceSynchronize();

	// copy result back to host
	checkErrorsCuda(hipMemcpy(positions, Device::positions,
		size * sizeof(glm::vec3),
		hipMemcpyDeviceToHost)
	);

}

////////////////////////////////////////////////////////////////////////////////////////////////////
// returns positions as flat array 
////////////////////////////////////////////////////////////////////////////////////////////////////
const float *
Cuda_Computing::getPositions() const {
	return glm::value_ptr(positions[0]);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// returns the number of bodies 
////////////////////////////////////////////////////////////////////////////////////////////////////
size_t 
Cuda_Computing::getSize() const {
	return size;
}



////////////////////////////////////////////////////////////////////////////////////////////////////
//! Entry point to device == KERNEL
////////////////////////////////////////////////////////////////////////////////////////////////////
